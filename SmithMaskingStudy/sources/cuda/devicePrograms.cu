#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018-2019 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include <optix_device.h>

#include <OptiX/LaunchParams.h>
#include <gdt/random/random.h>
#include <gdt/random/sampling.h>

/*! launch parameters in constant memory, filled in by optix upon
      optixLaunch (this gets filled in from the buffer we pass to
      optixLaunch) */
extern "C" __constant__ LaunchParams optixLaunchParams;

/*! per-ray data now captures random number generator, so programs
      can access RNG state */
struct PRD {
    Random<16> random;
    unsigned int seed;
    vec3sc color;
    bool backface;
    bool visible;
    bool onSurface;
};

static __forceinline__ __device__
void* unpackPointer(uint32_t i0, uint32_t i1)
{
    const uint64_t uptr = static_cast<uint64_t>(i0) << 32 | i1;
    void* ptr = reinterpret_cast<void*>(uptr);
    return ptr;
}

static __forceinline__ __device__
void  packPointer(void* ptr, uint32_t& i0, uint32_t& i1)
{
    const uint64_t uptr = reinterpret_cast<uint64_t>(ptr);
    i0 = uptr >> 32;
    i1 = uptr & 0x00000000ffffffff;
}

template<typename T>
static __forceinline__ __device__ T* getPRD()
{
    const uint32_t u0 = optixGetPayload_0();
    const uint32_t u1 = optixGetPayload_1();
    return reinterpret_cast<T*>(unpackPointer(u0, u1));
}

__forceinline__ __device__ vec3sc AABBIntersection(const box3sc& aabb, const vec3sc& O, const vec3sc& D)
{
    vec3sc v_tmin = vec3sc(
        (aabb.lower.x - O.x) / D.x,
        (aabb.lower.y - O.y) / D.y,
        (aabb.lower.z - O.z) / D.z
    );
    vec3sc v_tmax = vec3sc(
        (aabb.upper.x - O.x) / D.x,
        (aabb.upper.y - O.y) / D.y,
        (aabb.upper.z - O.z) / D.z
    );
    for (int i = 0; i < 3; ++i) {
        if (v_tmin[i] > v_tmax[i]) {
            scal tmp = v_tmin[i];
            v_tmin[i] = v_tmax[i];
            v_tmax[i] = tmp;
        }
    }
    scal t = min(v_tmax[0], min(v_tmax[1], v_tmax[2]));
    return O + t * D;
}

//------------------------------------------------------------------------------
// closest hit and anyhit programs for radiance-type rays.
//
// Note eventually we will have to create one pair of those for each
// ray type and each geometry type we want to render; but this
// simple example doesn't use any actual geometries yet, so we only
// create a single, dummy, set of them (we do have to have at least
// one group of them to set up the SBT)
//------------------------------------------------------------------------------

// closest hit program for a vertical ray launched somewhere above the surface.
// we want to find the point on the surface juste under the ray origin.
// we need to take care of the side effect and discard unreliable points.
extern "C" __global__ void __closesthit__radiance()
{
    // get the per-ray data structure reference :
    PRD& prd = *getPRD<PRD>();

    // get the mesh data :
    const TriangleMeshSBTData& sbtData
        = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();

    // get the triangle data :
    const int   primID = optixGetPrimitiveIndex(); // faceID
    const vec3i index = sbtData.index[primID];
    const vec3sc& A = sbtData.vertex[index.x];
    const vec3sc& B = sbtData.vertex[index.y];
    const vec3sc& C = sbtData.vertex[index.z];
    const scal u = optixGetTriangleBarycentrics().x;
    const scal v = optixGetTriangleBarycentrics().y;

    // check if face belongs to a set
    if (optixLaunchParams.sets.numberSets > 0) {
        prd.color = optixLaunchParams.sets.colorsBySet[optixLaunchParams.sets.colorsId[primID]];
    }

    // compute the geometric normal :
    vec3sc N = (optixLaunchParams.visibility.useSmooth && sbtData.normal) ?
        ((1.f - u - v) * sbtData.normal[index.x]
            + u * sbtData.normal[index.y]
            + v * sbtData.normal[index.z])
        : normalize(cross(B - A, C - A));
    if (N.z < 0.f) N = -N;

    vec3sc rayDir;
    unsigned int flags;
    unsigned int rayType;
    if (   optixLaunchParams.camera.programType == ProgramType::G1
        || optixLaunchParams.camera.programType == ProgramType::GAF)
    {
        rayDir = optixLaunchParams.visibility.directionOut;
        flags = OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT | OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT;
        rayType = VISIBILITY_RAY_TYPE;
    }
    else if (optixLaunchParams.camera.programType == ProgramType::AMBIENT_OCCLUSION)
    {
        scal r1 = prd.random.rnd(prd.seed); // [0, 1]
        scal r2 = prd.random.rnd(prd.seed); // [0, 1]
        vec3sc verticalDir = uniformHemisphereSampling(r1, r2);
        rayDir = rotateAlongNormal(verticalDir, N);

        flags = OPTIX_RAY_FLAG_NONE;
        rayType = AMBIENT_OCCLUSION_RAY_TYPE;
    }
    else
    {
        return;
    }

    // find the 3D point on the surface.
    const vec3sc surfPos = (1.f - u - v) * A + u * B + v * C;

    // If we are too close to the edge of the surface, discard the point for the minimal visibility
    // (if optixLaunchParams.sideEffect.borderPercentage > 0)
    if (sbtData.bounds.closest_distance(surfPos).x < optixLaunchParams.sideEffect.borderPercentage * sbtData.bounds.span().x / 2.f
        || sbtData.bounds.closest_distance(surfPos).y < optixLaunchParams.sideEffect.borderPercentage * sbtData.bounds.span().y / 2.f)
    {
        prd.onSurface = false;
        return;
    }
    prd.onSurface = true;

    if (dot(N, rayDir) <= 0) {
        prd.backface = true;
        return;
    }
    prd.backface = false;

    // pack the visibility boolean :
    uint32_t u0, u1;
    packPointer(&(prd.visible), u0, u1);

    // launch a ray in the visibility direction (OUTgoing) :
    optixTrace(optixLaunchParams.traversable,
        surfPos + (scal)1e-3 * N,
        rayDir,
        (scal)0.0, // tmin
        optixLaunchParams.visibility.tMax,  // tmax
        (scal)0.0,  // rayTime
        OptixVisibilityMask(255),
        flags,
        rayType,            // SBT offset
        RAY_TYPE_COUNT,     // SBT stride
        rayType,            // missSBTIndex 
        u0, u1);

    // If the point is not visible, then there is no possible mistake: the surface is hiding it.
    if (optixLaunchParams.sideEffect.directional && prd.visible)
    {
        vec3sc hit = AABBIntersection(sbtData.bounds, surfPos, -optixLaunchParams.visibility.directionOut);
        if (hit.z < sbtData.bounds.upper.z && hit.z > sbtData.bounds.lower.z) {
            prd.onSurface = false;
            return;
        }
    }

    if (prd.visible && optixLaunchParams.camera.programType == ProgramType::GAF) {
        rayDir = optixLaunchParams.visibility.directionIn;
        if (dot(N, rayDir) <= 0) {
            prd.visible = false;
            return;
        }
        optixTrace(optixLaunchParams.traversable,
            surfPos + (scal)1e-3 * N,
            rayDir,
            (scal)0.0, // tmin
            optixLaunchParams.visibility.tMax,  // tmax
            (scal)0.0,  // rayTime
            OptixVisibilityMask(255),
            flags,
            VISIBILITY_RAY_TYPE, // SBT offset
            RAY_TYPE_COUNT,      // SBT stride
            VISIBILITY_RAY_TYPE, // missSBTIndex 
            u0, u1);

        if (optixLaunchParams.sideEffect.directional && prd.visible)
        {
            vec3sc hit = AABBIntersection(sbtData.bounds, surfPos, -optixLaunchParams.visibility.directionIn);
            if (hit.z < sbtData.bounds.upper.z && hit.z > sbtData.bounds.lower.z) {
                prd.onSurface = false;
                return;
            }
        }
    }
}

// closest hit program vor a directional ray launched from the surface,
// with a specified tMax.
extern "C" __global__ void __closesthit__ambient_occlusion()
{
    // we hitted something, so the point is not visible
    bool& visibility = *getPRD<bool>();
    visibility = false;
}


//------------------------------------------------------------------------------
// any hit programs.
//------------------------------------------------------------------------------

extern "C" __global__ void __anyhit__visibility() {
    const TriangleMeshSBTData& sbtData = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();
    const vec3i index = sbtData.index[optixGetPrimitiveIndex()];
    const scal u = optixGetTriangleBarycentrics().x;
    const scal v = optixGetTriangleBarycentrics().y;
    const vec3sc hit = (1.f - u - v) * sbtData.vertex[index.x] + u * sbtData.vertex[index.y] + v * sbtData.vertex[index.z];

    bool& visibility = *getPRD<bool>();

    // If we are too close to the edge of the surface, consider no hit (visible)
    const scal epsilonX = 0.001 * sbtData.bounds.span().x;
    const scal epsilonY = 0.001 * sbtData.bounds.span().y;
    if (hit.x <= sbtData.bounds.lower.x + epsilonX || hit.x >= sbtData.bounds.upper.x - epsilonX
        || hit.y <= sbtData.bounds.lower.y + epsilonY || hit.y >= sbtData.bounds.upper.y - epsilonY)
    {
        visibility = true;
        return;
    }

    // we hitted something, so the point is not visible
    visibility = false;
}


//------------------------------------------------------------------------------
// miss programs.
// ------------------------------------------------------------------------------

extern "C" __global__ void __miss__radiance()
{
    PRD& prd = *getPRD<PRD>();
    prd.onSurface = false;
}

extern "C" __global__ void __miss__visibility()
{
    // we didn't hit anything, so the point is visible
    bool& visibility = *getPRD<bool>();
    visibility = true;
}



//------------------------------------------------------------------------------
// ray generation programs.
//------------------------------------------------------------------------------

extern "C" __global__ void __raygen__globalVisibility()
{
    // compute a test pattern based on pixel ID
    const scal ix = optixGetLaunchIndex().x;
    const scal iy = optixGetLaunchIndex().y;
    const int accumID = optixLaunchParams.frame.accumID;
    const auto& camera = optixLaunchParams.camera;

    // our per-ray data for this example. what we initialize it to
    // won't matter, since this value will be overwritten by either
    // the miss or hit program, anyway
    PRD prd;
    prd.seed = prd.random.init(ix + accumID * optixLaunchParams.frame.size.x,
        iy + accumID * optixLaunchParams.frame.size.y);
    prd.color = { 0.f, 0.f, 0.f };
    prd.backface = false;
    prd.visible = false;
    prd.onSurface = false;
    vec3sc pixelColorPRD{ 0.f, 0.f, 0.f };
    uint32_t validRays = 0;
    uint32_t visibleRays = 0;

    // the values we store the PRD pointer in:
    uint32_t u0, u1;
    packPointer(&prd, u0, u1);

    int numPixelSamples = optixLaunchParams.camera.nPixelSamples;

    for (int sampleID = 0; sampleID < numPixelSamples; sampleID++) {
        // normalized screen plane position, in [0,1]^2
        const float screen_u1 = prd.random.rnd(prd.seed);
        const float screen_u2 = prd.random.rnd(prd.seed);
        const vec2sc screen = vec2sc(ix + screen_u1, iy + screen_u2) / vec2sc(optixLaunchParams.frame.size);

        // generate ray origin (orthographic camera)
        vec3sc rayOrigin = camera.position
            + (screen.x - (scal)0.5) * camera.horizontal
            + (screen.y - (scal)0.5) * camera.vertical;

        optixTrace(optixLaunchParams.traversable,
            rayOrigin,
            optixLaunchParams.camera.direction, // ray direction
            (scal)0,    // tmin
            (scal)1e20,  // tmax
            (scal)0.0,   // rayTime
            OptixVisibilityMask(255),
            OPTIX_RAY_FLAG_DISABLE_ANYHIT,  // OPTIX_RAY_FLAG_NONE,
            RADIANCE_RAY_TYPE,            // SBT offset
            RAY_TYPE_COUNT,               // SBT stride
            RADIANCE_RAY_TYPE,            // missSBTIndex 
            u0, u1);

        if (optixLaunchParams.sets.numberSets > 0) {
            pixelColorPRD += prd.color;
        }
        else if (prd.onSurface && !prd.backface) {
            validRays++;
            if (prd.visible) {
                pixelColorPRD += vec3sc(0.f, 1.f, 0.f);
                visibleRays++;
            }
            else {
                pixelColorPRD += vec3sc(1.f, 0.f, 0.f);
            }
        }
        else {
            pixelColorPRD += vec3sc(0.f, 0.f, 1.f);
        }
    }

    pixelColorPRD /= numPixelSamples;
    int r, g, b;
    if (optixLaunchParams.camera.programType == ProgramType::AMBIENT_OCCLUSION) {
        r = g = b = int(255.99f * (scal)visibleRays / (scal)validRays);
    }
    else {
        r = int(255.99f * pixelColorPRD.x);
        g = int(255.99f * pixelColorPRD.y);
        b = int(255.99f * pixelColorPRD.z);
    }
    const uint32_t rgba = 0xff000000
        | (r << 0) | (g << 8) | (b << 16);


    // and write to frame buffer ...
    const uint32_t fbIndex = ix + iy * optixLaunchParams.frame.size.x;

    optixLaunchParams.frame.colorBuffer[fbIndex] = rgba;
    if (validRays == 0) {
        optixLaunchParams.frame.visibilityBuffer[fbIndex] = -1.f;
    }
    else {
        optixLaunchParams.frame.visibilityBuffer[fbIndex] = (scal)visibleRays / (scal)validRays;
    }
}